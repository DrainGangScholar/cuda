
#include <hip/hip_runtime.h>
#include <cassert>
#include <cmath>
#include <stdio.h>
#define N 1024
#define bytes N * sizeof(int)
__global__ void add(int *a, int *b, int *c) {
  int tid = threadIdx.x + (blockDim.x * blockIdx.x);
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}
int main() {
  int h_a[N], h_b[N], h_c[N];
  int *d_a;
  int *d_b;
  int *d_c;
  hipError_t cuda_status;
  cuda_status = hipMalloc(&d_a, bytes);
  assert(cuda_status == hipSuccess);
  cuda_status = hipMalloc(&d_b, bytes);
  assert(cuda_status == hipSuccess);
  cuda_status = hipMalloc(&d_c, bytes);
  assert(cuda_status == hipSuccess);

  for (int i = 0; i < N; i++) {
    h_a[i] = 1;
    h_b[i] = 2;
  }

  cuda_status = hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  assert(cuda_status == hipSuccess);
  cuda_status = hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
  assert(cuda_status == hipSuccess);

  int NUM_THREADS = 256;
  int NUM_BLOCKS = (int)ceil(N / NUM_THREADS);

  add<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c);
  cuda_status = hipGetLastError();
  assert(cuda_status == hipSuccess);

  cuda_status = hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
  assert(cuda_status == hipSuccess);

  for (int i = 0; i < N; i++) {
    assert(h_c[i] == h_a[i] + h_b[i]);
    printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
